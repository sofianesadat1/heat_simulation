#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "pngwriter.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16



struct pixel_t {
    uint8_t red;
    uint8_t green;
    uint8_t blue;
};


static int heat_colormap[256][3] = {
    { 59,  76,  192 }, { 59,  76,  192 }, { 60,  78,  194 }, { 61,  80,  195 },
    { 62,  81,  197 }, { 64,  83,  198 }, { 65,  85,  200 }, { 66,  87,  201 },
    { 67,  88,  203 }, { 68,  90,  204 }, { 69,  92,  206 }, { 71,  93,  207 },
    { 72,  95,  209 }, { 73,  97,  210 }, { 74,  99,  211 }, { 75,  100, 213 },
    { 77,  102, 214 }, { 78,  104, 215 }, { 79,  105, 217 }, { 80,  107, 218 },
    { 82,  109, 219 }, { 83,  110, 221 }, { 84,  112, 222 }, { 85,  114, 223 },
    { 87,  115, 224 }, { 88,  117, 225 }, { 89,  119, 227 }, { 90,  120, 228 },
    { 92,  122, 229 }, { 93,  124, 230 }, { 94,  125, 231 }, { 96,  127, 232 },
    { 97,  129, 233 }, { 98,  130, 234 }, { 100, 132, 235 }, { 101, 133, 236 },
    { 102, 135, 237 }, { 103, 137, 238 }, { 105, 138, 239 }, { 106, 140, 240 },
    { 107, 141, 240 }, { 109, 143, 241 }, { 110, 144, 242 }, { 111, 146, 243 },
    { 113, 147, 244 }, { 114, 149, 244 }, { 116, 150, 245 }, { 117, 152, 246 },
    { 118, 153, 246 }, { 120, 155, 247 }, { 121, 156, 248 }, { 122, 157, 248 },
    { 124, 159, 249 }, { 125, 160, 249 }, { 127, 162, 250 }, { 128, 163, 250 },
    { 129, 164, 251 }, { 131, 166, 251 }, { 132, 167, 252 }, { 133, 168, 252 },
    { 135, 170, 252 }, { 136, 171, 253 }, { 138, 172, 253 }, { 139, 174, 253 },
    { 140, 175, 254 }, { 142, 176, 254 }, { 143, 177, 254 }, { 145, 179, 254 },
    { 146, 180, 254 }, { 147, 181, 255 }, { 149, 182, 255 }, { 150, 183, 255 },
    { 152, 185, 255 }, { 153, 186, 255 }, { 154, 187, 255 }, { 156, 188, 255 },
    { 157, 189, 255 }, { 158, 190, 255 }, { 160, 191, 255 }, { 161, 192, 255 },
    { 163, 193, 255 }, { 164, 194, 254 }, { 165, 195, 254 }, { 167, 196, 254 },
    { 168, 197, 254 }, { 169, 198, 254 }, { 171, 199, 253 }, { 172, 200, 253 },
    { 173, 201, 253 }, { 175, 202, 252 }, { 176, 203, 252 }, { 177, 203, 252 },
    { 179, 204, 251 }, { 180, 205, 251 }, { 181, 206, 250 }, { 183, 207, 250 },
    { 184, 207, 249 }, { 185, 208, 249 }, { 186, 209, 248 }, { 188, 209, 247 },
    { 189, 210, 247 }, { 190, 211, 246 }, { 191, 211, 246 }, { 193, 212, 245 },
    { 194, 213, 244 }, { 195, 213, 243 }, { 196, 214, 243 }, { 198, 214, 242 },
    { 199, 215, 241 }, { 200, 215, 240 }, { 201, 216, 239 }, { 202, 216, 239 },
    { 204, 217, 238 }, { 205, 217, 237 }, { 206, 217, 236 }, { 207, 218, 235 },
    { 208, 218, 234 }, { 209, 218, 233 }, { 210, 219, 232 }, { 211, 219, 231 },
    { 212, 219, 230 }, { 214, 220, 229 }, { 215, 220, 228 }, { 216, 220, 227 },
    { 217, 220, 225 }, { 218, 220, 224 }, { 219, 220, 223 }, { 220, 221, 222 },
    { 221, 221, 221 }, { 222, 220, 219 }, { 223, 220, 218 }, { 224, 219, 216 },
    { 225, 219, 215 }, { 226, 218, 214 }, { 227, 218, 212 }, { 228, 217, 211 },
    { 229, 216, 209 }, { 230, 216, 208 }, { 231, 215, 206 }, { 232, 215, 205 },
    { 233, 214, 203 }, { 233, 213, 202 }, { 234, 212, 200 }, { 235, 212, 199 },
    { 236, 211, 197 }, { 237, 210, 196 }, { 237, 209, 194 }, { 238, 208, 193 },
    { 239, 208, 191 }, { 239, 207, 190 }, { 240, 206, 188 }, { 240, 205, 187 },
    { 241, 204, 185 }, { 242, 203, 183 }, { 242, 202, 182 }, { 243, 201, 180 },
    { 243, 200, 179 }, { 243, 199, 177 }, { 244, 198, 176 }, { 244, 197, 174 },
    { 245, 196, 173 }, { 245, 195, 171 }, { 245, 194, 169 }, { 246, 193, 168 },
    { 246, 192, 166 }, { 246, 190, 165 }, { 246, 189, 163 }, { 247, 188, 161 },
    { 247, 187, 160 }, { 247, 186, 158 }, { 247, 184, 157 }, { 247, 183, 155 },
    { 247, 182, 153 }, { 247, 181, 152 }, { 247, 179, 150 }, { 247, 178, 149 },
    { 247, 177, 147 }, { 247, 175, 146 }, { 247, 174, 144 }, { 247, 172, 142 },
    { 247, 171, 141 }, { 247, 170, 139 }, { 247, 168, 138 }, { 247, 167, 136 },
    { 247, 165, 135 }, { 246, 164, 133 }, { 246, 162, 131 }, { 246, 161, 130 },
    { 246, 159, 128 }, { 245, 158, 127 }, { 245, 156, 125 }, { 245, 155, 124 },
    { 244, 153, 122 }, { 244, 151, 121 }, { 243, 150, 119 }, { 243, 148, 117 },
    { 242, 147, 116 }, { 242, 145, 114 }, { 241, 143, 113 }, { 241, 142, 111 },
    { 240, 140, 110 }, { 240, 138, 108 }, { 239, 136, 107 }, { 239, 135, 105 },
    { 238, 133, 104 }, { 237, 131, 102 }, { 237, 129, 101 }, { 236, 128, 99  },
    { 235, 126, 98  }, { 235, 124, 96  }, { 234, 122, 95  }, { 233, 120, 94  },
    { 232, 118, 92  }, { 231, 117, 91  }, { 230, 115, 89  }, { 230, 113, 88  },
    { 229, 111, 86  }, { 228, 109, 85  }, { 227, 107, 84  }, { 226, 105, 82  },
    { 225, 103, 81  }, { 224, 101, 79  }, { 223, 99,  78  }, { 222, 97,  77  },
    { 221, 95,  75  }, { 220, 93,  74  }, { 219, 91,  73  }, { 218, 89,  71  },
    { 217, 87,  70  }, { 215, 85,  69  }, { 214, 82,  67  }, { 213, 80,  66  },
    { 212, 78,  65  }, { 211, 76,  64  }, { 210, 74,  62  }, { 208, 71,  61  },
    { 207, 69,  60  }, { 206, 67,  59  }, { 204, 64,  57  }, { 203, 62,  56  },
    { 202, 59,  55  }, { 200, 57,  54  }, { 199, 54,  53  }, { 198, 52,  51  },
    { 196, 49,  50  }, { 195, 46,  49  }, { 193, 43,  48  }, { 192, 40,  47  },
    { 191, 37,  46  }, { 189, 34,  44  }, { 188, 30,  43  }, { 186, 26,  42  },
    { 185, 22,  41  }, { 183, 17,  40  }, { 182, 11,  39  }, { 180, 4,   38  },
};

void cmap(double value, const double scaling, const double offset,
          pixel_t * pix)
{
    int ival;

    ival = (int) (value * scaling + offset);
    if (ival < 0) {             
        pix->red = 0;
        pix->green = 0;
        pix->blue = 255;
    } else if (ival > 255) {
        pix->red = 255;         
        pix->green = 0;
        pix->blue = 0;
    } else {
        pix->red = heat_colormap[ival][0];
        pix->green = heat_colormap[ival][1];
        pix->blue = heat_colormap[ival][2];
    }
}


int save_png(float *data, const int height, const int width, const char *fname,
             const char lang)
{
    FILE *fp;
    png_structp pngstruct_ptr = NULL;
    png_infop pnginfo_ptr = NULL;
    png_byte **row_pointers = NULL;
    int i, j;

    
    int status = -1;

    int pixel_size = 3;
    int depth = 8;

    fp = fopen(fname, "wb");
    if (fp == NULL)
        goto fopen_failed;

    pngstruct_ptr =
        png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

    if (pngstruct_ptr == NULL)
        goto pngstruct_create_failed;

    pnginfo_ptr = png_create_info_struct(pngstruct_ptr);

    if (pnginfo_ptr == NULL)
        goto pnginfo_create_failed;

    if (setjmp(png_jmpbuf(pngstruct_ptr)))
        goto setjmp_failed;

    png_set_IHDR(pngstruct_ptr, pnginfo_ptr, (size_t) width,
                 (size_t) height, depth, PNG_COLOR_TYPE_RGB,
                 PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);

    row_pointers = (png_bytep*) png_malloc(pngstruct_ptr, height * sizeof(png_byte *));

    for (i = 0; i < height; i++) {
        png_bytep row = (png_bytep) png_malloc(pngstruct_ptr, sizeof(uint8_t) * width * pixel_size);

        row_pointers[i] = row;

        
        if (lang == 'c' || lang == 'C') {
            for (j = 0; j < width; j++) {
                pixel_t pixel;
                
                cmap(data[j + i * width], 2.55, 0.0, &pixel);
                *row++ = pixel.red;
                *row++ = pixel.green;
                *row++ = pixel.blue;
            }
        } else {
            for (j = 0; j < width; j++) {
                pixel_t pixel;
                
                cmap(data[i + j * height], 2.55, 0.0, &pixel);
                *row++ = pixel.red;
                *row++ = pixel.green;
                *row++ = pixel.blue;
            }
        }
    }

    png_init_io(pngstruct_ptr, fp);
    png_set_rows(pngstruct_ptr, pnginfo_ptr, row_pointers);
    png_write_png(pngstruct_ptr, pnginfo_ptr,
                  PNG_TRANSFORM_IDENTITY, NULL);

    status = 0;

    for (i = 0; i < height; i++) {
        png_free(pngstruct_ptr, row_pointers[i]);
    }
    png_free(pngstruct_ptr, row_pointers);

  setjmp_failed:
  pnginfo_create_failed:
    png_destroy_write_struct(&pngstruct_ptr, &pnginfo_ptr);
  pngstruct_create_failed:
    fclose(fp);
  fopen_failed:
    return status;
}



int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i*width + j;
}

__global__ void evolve_kernel(const float* Un, float* Unp1, const int nx, const int ny, const float dx2, const float dy2, const float aTimesDt)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i > 0 && i < nx - 1)
    {
        int j = threadIdx.y + blockIdx.y*blockDim.y;
        if (j > 0 && j < ny - 1)
        {
            const int index = getIndex(i, j, ny);
            float uij = Un[index];
            float uim1j = Un[getIndex(i-1, j, ny)];
            float uijm1 = Un[getIndex(i, j-1, ny)];
            float uip1j = Un[getIndex(i+1, j, ny)];
            float uijp1 = Un[getIndex(i, j+1, ny)];

            // Explicit scheme
            Unp1[index] = uij + aTimesDt * ( (uim1j - 2.0*uij + uip1j)/dx2 + (uijm1 - 2.0*uij + uijp1)/dy2 );
        }
    }
}

int main()
{   int e = 7500;
    const int nx = e;   // Width of the area
    const int ny = e;   // Height of the area

    const float a = 0.5;     // Diffusion constant

    const float dx = 0.01;   // Horizontal grid spacing 
    const float dy = 0.01;   // Vertical grid spacing

    const float dx2 = dx*dx;
    const float dy2 = dy*dy;

    const float dt = dx2 * dy2 / (2.0 * a * (dx2 + dy2)); // Largest stable time step
    const int numSteps = 5000;                             // Number of time steps
    const int outputEvery = 5000;                          // How frequently to write output image

    int numElements = nx*ny;

    
    float* h_Un   = (float*)calloc(numElements, sizeof(float));

    
    float radius2 = (nx/6.0) * (nx/6.0);
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            int index = getIndex(i, j, ny);
            
            float ds2 = (i - nx/2) * (i - nx/2) + (j - ny/2)*(j - ny/2);
            if (ds2 < radius2)
            {
                h_Un[index] = 65.0;
            }
            else
            {
                h_Un[index] = 5.0;
            }
        }
    }

    float* d_Un;
    float* d_Unp1;

    hipMalloc((void**)&d_Un, numElements*sizeof(float));
    hipMalloc((void**)&d_Unp1, numElements*sizeof(float));

    hipMemcpy(d_Un, h_Un, numElements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Unp1, h_Un, numElements*sizeof(float), hipMemcpyHostToDevice);

    dim3 numBlocks(nx/BLOCK_SIZE_X + 1, ny/BLOCK_SIZE_Y + 1);
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    
    clock_t start = clock();

    
    for (int n = 0; n <= numSteps; n++)
    {
        evolve_kernel<<<numBlocks, threadsPerBlock>>>(d_Un, d_Unp1, nx, ny, dx2, dy2, a*dt);

        
        if (n % outputEvery == 0)
        {
            hipMemcpy(h_Un, d_Un, numElements*sizeof(float), hipMemcpyDeviceToHost);
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                exit(0);
            }
            char filename[64];
            sprintf(filename, "heat_%04d.png", n);
            save_png(h_Un, nx, ny, filename, 'c');
        }

        std::swap(d_Un, d_Unp1);
    }

    
    clock_t finish = clock();
    printf("It took %f seconds\n", (double)(finish - start) / CLOCKS_PER_SEC);

    
    free(h_Un);

    hipFree(d_Un);
    hipFree(d_Unp1);
    
    return 0;
}